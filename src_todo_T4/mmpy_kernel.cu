#include "hip/hip_runtime.h"
// ;-*- mode: c;-*-
// Matrix multiply device code
#include "../src/types.h"
#include "../src/utils.h"
#include "mytypes.h"
#include <assert.h>
#include <math.h>
using namespace std;

#include <stdio.h>

#ifdef NAIVE
__global__ void matMul(int N, _FTYPE_ *C, _FTYPE_ *A, _FTYPE_ *B) {

  int I = blockIdx.y * blockDim.y + threadIdx.y;
  int J = blockIdx.x * blockDim.x + threadIdx.x;

  if ((I < N) && (J < N)) {
    _FTYPE_ _c = 0;
    for (unsigned int k = 0; k < N; k++) {
      _FTYPE_ a = A[I * N + k];
      _FTYPE_ b = B[k * N + J];
      _c += a * b;
    }
    C[I * N + J] = _c;
  }
}

#else
// You should be changing the kernel here for the non naive implementation.
__global__ void matMul(int N, _FTYPE_ *C, _FTYPE_ *A, _FTYPE_ *B) {
  __shared__ double As[TILEDIM_M][TILEDIM_K], Bs[TILEDIM_K][TILEDIM_N];

  int ty = threadIdx.y, tx = threadIdx.x;
  int by = blockIdx.y, bx = blockIdx.x;

  int I = by * TILEDIM_K + ty;
  int J = bx * TILEDIM_K + tx;

  double Cij[2] = {0, 0};
  for (int kk = 0; kk < (N / TILEDIM_M + (N % TILEDIM_M != 0)); kk++) {
    if (I < N && kk * TILEDIM_M + tx < N)
      As[ty][tx] = A[I * N + kk * TILEDIM_M + tx];
    else
      As[ty][tx] = 0;

    if ((I + TILEDIM_N / TILESCALE_N) < N && kk * TILEDIM_M + tx < N)
      As[ty + TILEDIM_N / TILESCALE_N][tx] =
          A[(I + TILEDIM_N / TILESCALE_N) * N + kk * TILEDIM_M + tx];
    else
      As[ty + TILEDIM_N / TILESCALE_N][tx] = 0;

    if (kk * TILEDIM_M + ty < N && J < N)
      Bs[ty][tx] = B[(kk * TILEDIM_M + ty) * N + J];
    else
      Bs[ty][tx] = 0;

    if (kk * TILEDIM_M + ty + TILEDIM_N / TILESCALE_N < N && J < N)
      Bs[ty + TILEDIM_N / TILESCALE_N][tx] =
          B[(kk * TILEDIM_M + ty + TILEDIM_N / TILESCALE_N) * N + J];
    else
      Bs[ty + TILEDIM_N / TILESCALE_N][tx] = 0;

    __syncthreads();
#pragma unroll
    for (int k = 0; k < TILEDIM_K; k++) {
      Cij[0] += As[ty][k] * Bs[k][tx];
      Cij[1] += As[ty + TILEDIM_N / TILESCALE_N][k] * Bs[k][tx];
    }
    __syncthreads();
  }

  if (I < N && J < N)
    C[I * N + J] = Cij[0];
  if (I + TILEDIM_N / TILESCALE_N < N && J < N)
    C[(I + TILEDIM_N / TILESCALE_N) * N + J] = Cij[1];
}
#endif
